#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

/* Define the matrix size */
#define NX 200
#define NY 100

/* Size of a block */
#define BLOCKSIZE 256

__global__ void kernadd (float* mout, float* min1, float *min2, int nx, int ny)
{
  int i, j, index;
  index = blockDim.x*blockIdx.x+threadIdx.x;
  j = index/nx;
  i = index - j*nx;
  if ((i < nx) && (j < ny))
    mout[index] = min1[index] + min2[index];
    
}


/*******************************************************/
/*  We initialize the vectors with random values       */
/*******************************************************/

void Init(float* mat, int nx, int ny) {
  int i, j;
  for (i = 0; i < nx; i++) {	/* 2D loop */
    for (j = 0; j < ny; j++) {
      mat[i+j*nx] = drand48 ();	/* position of cell (i,j) */
    }
  }
}


/*******************************************************/
/*            MAIN PROGRAM                             */
/*******************************************************/

int main () {
  int i=0, error=0, nx=NX, ny=NY;
  float diff;

  /* Matrix allocation */
  float *mat_in1 = (float*) malloc(nx * ny * sizeof(float));
  float *mat_in2 = (float*) malloc(nx * ny * sizeof(float));
  float *mat_out = (float*) malloc(nx * ny * sizeof(float));

  /* Matrix allocation on device */
  float *mat_out_gpu, *mat_in1_gpu, *mat_in2_gpu;
  /* TO DO : do the allocation below, using hipMalloc()*/
  

  /* Matrix initialization */
  Init(mat_in1, nx, ny);
  Init(mat_in2, nx, ny);  
  
  /* TO DO : write below the instructions to copy it to the device */

  
  /* TO DO : complete the number of blocks below */
  int numBlocks = ...;
 
  /* TO DO : kernel invocation */
  
  
  hipDeviceSynchronize();
  
  /* We now transfer back the matrix from the device to the host */
  /* TO DO : write hipMemcpy() instruction below */
  
    
  /* free memory */
  hipFree(mat_out_gpu);
  hipFree(mat_in1_gpu);
  hipFree(mat_in2_gpu);

  /* We now check that the result is correct */

  for (i=0; i< nx*ny; i++) {	/* No need for a 2D loop, actually ! */
    diff = mat_out[i] - (mat_in1[i]+mat_in2[i]);
    if (fabs(diff) > 0.0000001f) {
      error = 1;
    }
  }

  if (error) {
    printf("FAILED\n");
  }
  else {
    printf("PASSED\n");
  }
  free (mat_in1);
  free (mat_in2);
  free (mat_out);
}



