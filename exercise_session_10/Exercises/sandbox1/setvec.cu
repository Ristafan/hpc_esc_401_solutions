#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define BLOCKSIZE 256

__global__ void kern_set_val (float *gpu_ptr, float value, int n) {
  int i;
  //TO DO: evaluate the value of i 
  gpu_ptr[i] = value;
}

int main () {
  int i, failed=0;
  int N = 1024;   // size of vector
  float value = 11.0; //value ptr should be initialized to
  float *ptr;     // Host pointer 
  float *gpu_ptr; // Device pointer
  
  /* Allocate vector in Host*/
  ptr = (float *)malloc(sizeof(float)*N);
  /* Allocate vector in Device*/
  hipMalloc (&gpu_ptr, sizeof(float)*N);

  //TO DO : write kernel invocation here

  hipDeviceSynchronize ();

  //TO DO : copy data to host

  hipFree (gpu_ptr);

  /* Now check that it did what we want */

  for (i = 0; i < 10; i++)//first ten values are written
    printf ("%f\t", ptr[i]);
  printf ("\n");
  for (i = N-10; i < N; i++)//last ten values are written
    printf ("%f\t", ptr[i]);
  printf ("\n");

  for (i = 0; i < N; i++) {//All values are compared
    if (fabs(ptr[i]-value) > 1e-8) {
      failed=1;
    }
  }
  if (failed) {
    printf ("FAILED !!\n");
  } else {
    printf ("PASSED !!\n");
  }    
  free (ptr);
}
