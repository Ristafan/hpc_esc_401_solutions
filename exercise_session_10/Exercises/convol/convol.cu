#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define STOP 0
#define START 1


extern "C" void chrono (int kind, float *time);

__global__ void kconvol (float *gpu_a, float *gpu_b, int n) {
  int i, j, l;
  // TO DO : evaluate the global 1D index l of the current thread,
  // using blockDim, blockIdx and threadIdx.
  l = blockDim.x*blockIdx.x+threadIdx.x;
  
  // TO DO : evaluate global indeces of thread (i,j) from the index l
  j = l / n;
  i = l % n;

  if ((i >= n) || (j >= n)) return;
  if ((i == 0) || (j == 0) || (i == n-1) || (j == n-1))  {
    gpu_b[l] = gpu_a[l]; // edges are untouched
  }
  else
    // TO DO : fill up the MISSING indices below
    gpu_b[l]=(1./5.)*(gpu_a[l-n] + gpu_a[l-1] + gpu_a[l] + gpu_a[l+1]+ gpu_a[l+n]);
}

extern "C" void gpu_convol (float *a, float *b, int n, int blockSize) {
  float *gpu_a;
  float *gpu_b;
  hipError_t err;
  float time;
  
  err = hipMalloc (&gpu_a, n*n*sizeof(float));
  if (err != 0) {
    printf ("Error allocating gpu_a: %s\n", hipGetErrorString (err));
    exit (1);
  }
  err = hipMalloc (&gpu_b, n*n*sizeof(float));
  if (err != 0) {
    printf ("Error allocating gpu_b: %s\n", hipGetErrorString (err));
    exit (1);
  }
  
  hipMemcpy (gpu_a, a, n*n*sizeof(float), hipMemcpyHostToDevice);
    
  // NOTE : the chronometer below does not contemplate overhead of memory allocation and
  // memory transfer.
  chrono (START, &time);
  // TO DO : the number of blocks is missing below in the kernel invocation
  kconvol <<<((n*n) + blockSize - 1)/blockSize,blockSize>>> (gpu_a, gpu_b, n);
  err=hipDeviceSynchronize ();
  chrono (STOP, &time);
  printf ("Convolution took  %f sec. on GPU\n", time);
  hipMemcpy (b, gpu_b, n*n*sizeof(float), hipMemcpyDeviceToHost);
  if (err != 0) {
    printf ("%s\n", hipGetErrorString (err));
    exit (1);
  }
  hipFree (gpu_a);
  hipFree (gpu_b);
}

