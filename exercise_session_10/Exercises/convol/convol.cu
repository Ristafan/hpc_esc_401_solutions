#include "hip/hip_runtime.h"
#include <time.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#define STOP 0
#define START 1

#define BLOCKSIZE 256


extern "C" void chrono (int kind, float *time);

__global__ void kconvol (float *gpu_a, float *gpu_b, int n) {
  int i, j, l;
  // TO DO : evaluate the global 1D index l of the current thread,
  // using blockDim, blockIdx and threadIdx.

  
  // TO DO : evaluate global indeces of thread (i,j) from the index l
  j = ...;
  i = ...;

  if ((i >= n) || (j >= n)) return;
  if ((i == 0) || (j == 0) || (i == n-1) || (j == n-1))  {
    gpu_b[l] = gpu_a[l]; // edges are untouched
  }
  else
    // TO DO : fill up the MISSING indices below
    gpu_b[l]=(1./5.)*(gpu_a[l-n] + gpu_a[/*MISSING*/] + gpu_a[l] + gpu_a[l+1]+ gpu_a[/*MISSING*/]);
}

extern "C" void gpu_convol (float *a, float *b, int n) {
  float *gpu_a;
  float *gpu_b;
  hipError_t err;
  float time;
  
  err = hipMalloc (&gpu_a, n*n*sizeof(float));
  if (err != 0) {
    printf ("Error allocating gpu_a: %s\n", hipGetErrorString (err));
    exit (1);
  }
  err = hipMalloc (&gpu_b, n*n*sizeof(float));
  if (err != 0) {
    printf ("Error allocating gpu_b: %s\n", hipGetErrorString (err));
    exit (1);
  }
  
  hipMemcpy (gpu_a, a, n*n*sizeof(float), hipMemcpyHostToDevice);
    
  // NOTE : the chronometer below does not contemplate overhead of memory allocation and
  // memory transfer.
  chrono (START, &time);
  // TO DO : the number of blocks is missing below in the kernel invocation
  kconvol <<</*MISSING*/,BLOCKSIZE>>> (gpu_a, gpu_b, n);
  err=hipDeviceSynchronize ();
  chrono (STOP, &time);
  printf ("Convolution took  %f sec. on GPU\n", time);
  hipMemcpy (b, gpu_b, n*n*sizeof(float), hipMemcpyDeviceToHost);
  if (err != 0) {
    printf ("%s\n", hipGetErrorString (err));
    exit (1);
  }
  hipFree (gpu_a);
  hipFree (gpu_b);
}
