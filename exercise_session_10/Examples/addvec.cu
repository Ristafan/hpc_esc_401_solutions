#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
	int index = blockIdx.x;
	c[index] = a[index] + b[index];
}

#define N 512

void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%10;
	}
}

int main(void) {
	int *a, *b, *c;	// host copies of a, b, c
	int *a_d, *b_d, *c_d;	// device copies of a, b, c
	int size = N * sizeof(int);
	
	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&a_d, size);
	hipMalloc((void **)&b_d, size);
	hipMalloc((void **)&c_d, size);

	// Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size); random_ints(a, N);
	b = (int *)malloc(size); random_ints(b, N);
	c = (int *)malloc(size);

	// Copy inputs to device
	hipMemcpy(a_d, a, size, hipMemcpyHostToDevice);
	hipMemcpy(b_d, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<N,1>>>(a_d, b_d, c_d);
	
	// Copy result back to host
	hipMemcpy(c, c_d, size, hipMemcpyDeviceToHost);
		
    // Print results
	for(int i=0; i<N; i++)printf("%d) %d + %d = %d\n",i,a[i],b[i],c[i]);
        	
    // Cleanup
    free(a); free(b); free(c);
	hipFree(a_d); hipFree(b_d); hipFree(c_d);
	return 0;
}
