#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c, int n) {
	int index = threadIdx.x+blockIdx.x*blockDim.x;
	if(index < n)
		c[index] = a[index] + b[index];
}

#define N 9
#define M 5

void random_ints(int* x, int size)
{
	int i;
	for (i=0;i<size;i++) {
		x[i]=rand()%10;
	}
}

int main(void) {
	int *a1, *b1, *c1, *a2, *b2, *c2;	// host copies of a, b, c
	int *a1_d, *b1_d, *c1_d, *a2_d, *b2_d, *c2_d;	// device copies of a, b, c
	int size = N * sizeof(int);
    
    hipStream_t stream1,stream2;
    hipError_t  error;
    error = hipStreamCreate(&stream1);
    error = hipStreamCreate(&stream2);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&a1_d, size);
	hipMalloc((void **)&b1_d, size);
    hipMalloc((void **)&c1_d, size);
    hipMalloc((void **)&a2_d, size);
	hipMalloc((void **)&b2_d, size);
	hipMalloc((void **)&c2_d, size);

	// Alloc space for host copies of a, b, c and setup input values
	a1 = (int *)malloc(size); random_ints(a1, N);
    b1 = (int *)malloc(size); random_ints(b1, N);
    c1 = (int *)malloc(size);
    a2 = (int *)malloc(size); random_ints(a2, N);
    b2 = (int *)malloc(size); random_ints(b2, N);
    c2 = (int *)malloc(size);


	// Copy inputs to device
	hipMemcpyAsync(a1_d, a1, size, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(b1_d, b1, size, hipMemcpyHostToDevice, stream1);
    
    hipMemcpyAsync(a2_d, a2, size, hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(b2_d, b2, size, hipMemcpyHostToDevice, stream2);

	// Launch add() kernel on GPU
    add<<<(N+M-1)/M,M,0,stream1>>>(a1_d, b1_d, c1_d,N);
    add<<<(N+M-1)/M,M,0,stream2>>>(a2_d, b2_d, c2_d,N);

	// Copy result back to host
    hipMemcpyAsync(c1, c1_d, size, hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(c2, c2_d, size, hipMemcpyDeviceToHost, stream2);
        
    /*error=cudaStreamSynchronize(stream1);
    // Print results
    for(int i=0; i<N; i++)printf("stream1 %d + %d = %d\n",a1[i],b1[i],c1[i]);
    
    error=cudaStreamSynchronize(stream2);
    // Print results
	for(int i=0; i<N; i++)printf("stream2 %d + %d = %d\n",a2[i],b2[i],c2[i]);
    */

    hipDeviceSynchronize();
    for(int i=0; i<N; i++)printf("stream1 %d + %d = %d\t stream2 %d + %d = %d\n",a1[i],b1[i],c1[i],a2[i],b2[i],c2[i]);
    if (error != 0) {
        printf ("%s\n", hipGetErrorString (error));
        exit (1);
    }
        	
    // Cleanup
    free(a1); free(b1); free(c1);
    free(a2); free(b2); free(c2);
    hipFree(a1_d); hipFree(b1_d); hipFree(c1_d);
    hipFree(a2_d); hipFree(b2_d); hipFree(c2_d);
	return 0;
}
