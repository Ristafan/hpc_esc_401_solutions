#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main(void) {
    int a, b, c;	            // host copies of a, b, c
    int *a_d, *b_d, *c_d;	    // device copies of a, b, c
    int size = sizeof(int);
    
    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&a_d, size);
    hipMalloc((void **)&b_d, size);
    hipMalloc((void **)&c_d, size);

    // Setup input values
    a = 2;
    b = 7;

	// Copy inputs to device
	hipMemcpy(a_d, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(b_d, &b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<1,1>>>(a_d, b_d, c_d);

	// Copy result back to host
	hipMemcpy(&c, c_d, size, hipMemcpyDeviceToHost);
		
	// Print the result
	printf("c = %d\n",c);
		
	// Cleanup
	hipFree(a_d); hipFree(b_d); hipFree(c_d);
	return 0;
}
