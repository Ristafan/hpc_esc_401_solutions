// Using CUDA device to calculate pi
#include <stdio.h>
#include <hip/hip_runtime.h>
extern "C" double getTime(void);

#define NBIN 1000000000  // Number of bins
#define NUM_BLOCK  (2*56)  // Number of thread blocks
#define NUM_THREAD  (2*8)  // Number of threads per block

// Kernel that executes on the CUDA device
__global__ void cal_pi(double *sum, int nbin, double step, int nthreads, int nblocks) {
	int i;
	double x;
	int idx = blockIdx.x*blockDim.x+threadIdx.x;  // Sequential thread index across the blocks
	for (i=idx; i< nbin; i+=nthreads*nblocks) {
		x = (i+0.5)*step;
		sum[idx] += 4.0/(1.0+x*x);
	}
}

// Main routine that executes on the host
int main(void) {
	dim3 dimGrid(NUM_BLOCK,1,1);  // Grid dimensions
	dim3 dimBlock(NUM_THREAD,1,1);  // Block dimensions
	double *sumHost, *sumDev;  // Pointer to host & device arrays
	double pi = 0;
	int tid;

	double step = 1.0/NBIN;  // Step size
	size_t size = NUM_BLOCK*NUM_THREAD*sizeof(double);  //Array memory size
	sumHost = (double *)malloc(size);  //  Allocate array on host
	hipMalloc((void **) &sumDev, size);  // Allocate array on device
   	double start = getTime();

	// Initialize array in device to 0
	hipMemset(sumDev, 0, size);
	// Do calculation on device
	cal_pi <<<dimGrid, dimBlock>>> (sumDev, NBIN, step, NUM_THREAD, NUM_BLOCK); // call CUDA kernel
	// Retrieve result from device and store it in host array
	hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);
	for(tid=0; tid<NUM_THREAD*NUM_BLOCK; tid++)
		pi += sumHost[tid];
	pi *= step;

	// Print results
	double delta = getTime() - start;
	printf("PI = %.16g computed in %.4g seconds\n", pi, delta);
	// Cleanup
	free(sumHost);
	hipFree(sumDev);

	return 0;
}

