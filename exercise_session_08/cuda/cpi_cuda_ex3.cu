
// Using CUDA device to calculate pi
#include <stdio.h>
#include <hip/hip_runtime.h>
extern "C" double getTime(void);

#define NBIN 1000000000  // Number of bins

// Kernel that executes on the CUDA device
__global__ void cal_pi(double *sum, int nbin, double step, int nthreads, int nblocks) {
    int i;
    double x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Sequential thread index across the blocks
    for (i = idx; i < nbin; i += nthreads * nblocks) {
        x = (i + 0.5) * step;
        sum[idx] += 4.0 / (1.0 + x * x);
    }
}

// Main routine that executes on the host
int main(void) {
    int num_blocks[] = {60, 120, 180, 240, 300, 360, 420, 600};
    int num_threads[] = {16, 32, 48, 64, 80, 96, 112, 128, 144, 160};

    double *sumHost, *sumDev;  // Pointer to host & device arrays
    double pi = 0;
    double step = 1.0 / NBIN;  // Step size

    for (int b = 0; b < sizeof(num_blocks) / sizeof(num_blocks[0]); b++) {
        for (int t = 0; t < sizeof(num_threads) / sizeof(num_threads[0]); t++) {
            int NUM_BLOCK = num_blocks[b];
            int NUM_THREAD = num_threads[t];

            dim3 dimGrid(NUM_BLOCK, 1, 1);   // Grid dimensions
            dim3 dimBlock(NUM_THREAD, 1, 1); // Block dimensions

            size_t size = NUM_BLOCK * NUM_THREAD * sizeof(double);  // Array memory size
            sumHost = (double *)malloc(size);  // Allocate array on host
            hipMalloc((void **)&sumDev, size);  // Allocate array on device

            double start = getTime();

            // Initialize array in device to 0
            hipMemset(sumDev, 0, size);

            // Do calculation on device
            cal_pi<<<dimGrid, dimBlock>>>(sumDev, NBIN, step, NUM_THREAD, NUM_BLOCK); // call CUDA kernel

            // Retrieve result from device and store it in host array
            hipMemcpy(sumHost, sumDev, size, hipMemcpyDeviceToHost);

            pi = 0.0;
            for (int tid = 0; tid < NUM_THREAD * NUM_BLOCK; tid++) {
                pi += sumHost[tid];
            }
            pi *= step;

            double delta = getTime() - start;
            printf("NUM_BLOCK=%d, NUM_THREAD=%d: PI = %.16g computed in %.4g seconds\n", NUM_BLOCK, NUM_THREAD, pi, delta);

            //Cleanup
            free(sumHost);
            hipFree(sumDev);
        }
    }

    return 0;
}

